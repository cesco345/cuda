// A simple Hello World CUDA program.

// #include the entire body of the cuPrintf code
//#include "util/cuPrintf.cu"

// #include <stdio.h> for host printf

#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void device_greetings(void)
{
   printf("Hello, world from the device!\n");
}


int main(void)
{
 // greet from the host
   printf("Hello, world from the host!\n");

   // initialize cuPrintf
   //cudaPrintfInit();

   // launch a kernel with a single thread to greet from th>
   device_greetings<<<1,1>>>();

  // display the device's greeting
   hipDeviceSynchronize();

   //cudaDeviceReset;
   return 0;
 }
