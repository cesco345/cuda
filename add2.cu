
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

__global__ 
void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}


int main(){

    // create the arrays that will hold the data in the CPU
    int a[N], b[N], c[N];    

    // create the pointers that will hold the data in the GPU
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory to the GPU
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i<N ; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy memory from host to device (GPU)
    hipMemcpy(dev_a, a, (N) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, (N) * sizeof(int), hipMemcpyHostToDevice);
    
    add<<< N, 1 >>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // display the results
    for(int i=0; i<N; i++)
    {
        printf("%d + %d = %d \n", a[i], b[i], c[i]);
    }

    hipFree(dev_a); 
    hipFree(dev_b); 
    hipFree(dev_c);

    printf("\n");

    return 0;
}
