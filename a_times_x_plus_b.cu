#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x+blockDim.x+threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}

int main (void)
{
    int N = 1 << 10;
    float *h_x, *h_y, *d_x, *d_y;
    
    h_x = (float*)malloc(N*sizeof(float));
    h_y = (float*)malloc(N*sizeof(float));
    
    hipMalloc(&d_x, N*sizeof(float));
    hipMalloc(&d_y, N*sizeof(float));
    
    for (int i = 0; i < N; i++)
    {
        h_x[i] = 1.0f;
        h_y[i] = 2.0f;
    }
    
    hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice);
    
    //perform SAXPY on 1M elements
    saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    
    hipMemcpy(h_y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);
    
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = max(maxError, abs(h_y[i]-4.0f));
    printf("Max error: %f\n", maxError);
     
    hipFree(d_x);
    hipFree(d_y);
    free(h_x);
    free(h_y);
}
