#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void countThreads(void){
           int block_id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

           int block_offset = block_id * blockDim.x * blockDim.y * blockDim.z;

/* int block_id =
         blockIdx.x +
         blockIdx.y * gridDim.x +
         blockIdx.z * gridDim.x * gridDim.y;

           int block_offset = 
                block_id *
                blockDim.x * blockDim.y * blockDim.z;
*/

}


int main( int argc, char **argv)
{
        int thread_offset =  threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
        int id = block_offset + thread_offset;

        const int b_x = 2, b_y = 3, b_z = 4;
        const int t_x = 3, t_y = 3, t_z = 3;

        int blocks_per_grid = b_x * b_y * b_z;
        int threads_per_block = t_x * t_y * t_z;

        printf("%d blocs per grid\n", blocks_per_grid);
        printf("%d threads per block\n", threads_per_block);
        printf("%d total threads\n", blocks_per_grid * threads_per_block);

        dim3 blocksPerGrid(b_x, b_y, b_z);
        dim3 threadsPerBlock(t_x, t_y, t_z);

        countThreads<<<blocksPerGrid, threadsPerBlock>>>();
        hipDeviceSynchronize();

        printf("%04d    |    Block(%d %d %d)   =   %3d    |     Thread(%d %d %d)   = %3d\n",
            id,
            blockIdx.x, blockIdx.y, blockIdx.z, block_offset, 
            threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
        return 0 ;

}
