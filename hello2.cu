#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void hello_cuda()
{
	printf("Hello CUDA world \n");
}

int main()
{
	int nx, ny;
	nx = 2;
	ny = 2;

	dim3 block(2, 1);
	dim3 grid(nx / block.x,ny / block.y);

	hello_cuda << < 2,2 >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

